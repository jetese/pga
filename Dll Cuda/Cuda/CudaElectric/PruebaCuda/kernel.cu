#include "hip/hip_runtime.h"
﻿#include "kernel.h"
#include <stdio.h>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include "kernel.h"

#define SIZE 256
#define XSIZE (160)
#define YSIZE (400)
#define ZSIZE (1601)
//float voltajefinal[ZSIZE*YSIZE*XSIZE];


__global__ void initvolt(int *tamx, int *tamy, int *tamz, float* volt) {
	int posx = threadIdx.x + blockIdx.x * blockDim.x;

	if (posx < *tamz * *tamy * *tamx) {
		volt[posx] = 0.9;
	}
}

__global__ void module(int* numElectrodos, int *division, int * resto, int *indiceEgm, int * numTiempo, float *contentEgm, int * indexx, int * indexy, float * minValue, float * maxValue, float * auxLerp, float * maxScale, float * minScale, float* volt, int *tamx, int *tamy, int *tamz, int* indexvolt) {
	int pos = threadIdx.x + blockIdx.x * blockDim.x;
	if (pos < 88055) {
		resto[pos] = pos % *numElectrodos;
		division[pos] = pos / *numElectrodos;
		//x + WIDTH * (y + DEPTH * z)
		indiceEgm[pos] = resto[pos] * *numTiempo + division[pos];
		indexvolt[pos] = indexx[resto[pos]] + *tamx * (indexy[resto[pos]] + division[pos] * *tamy);
		
		auxLerp[pos] = (contentEgm[indiceEgm[pos]] + (-*minValue)) / (*maxValue + (-*minValue));
		volt[indexvolt[pos]] = (1 - auxLerp[pos]) * *minScale + auxLerp[pos] * *maxScale;
	}
}


#define TAM (88064) //Necesitamos un poco as porque trunca
#define THREADS_PER_BLOCK 256
#define TAM2 (102464000)
#define THREADS_PER_BLOCK2 1024
//int main(void) {
float* kernelArritmias(int numeroElectrodos, int sizePintado, int numeroTiempo, int sizeVoltaje, float * contenidoEgm, int *indicex, int *indicey, float valorMinimo, float valorMaximo, float escalaMaxima, float escalaMinima, float * prueba) {
	//Declaraci�n de variables
	float * voltajefinal;
	float *voltaje;
	int *indexvolt;//, indiceegmlol[88055];
	int xtam = XSIZE, ytam = YSIZE, ztam = ZSIZE;
	int * dev_xtam, *dev_ytam, *dev_ztam;
	float *dev_volt;

	int *dev_numeroElectrodos, *dev_division, *dev_resto, *dev_indiceEgm, *dev_numeroTiempo, *dev_indicex, *dev_indicey, *dev_indexvolt;
	float  *dev_contenidoEgm, *dev_valorMinimo, *dev_valorMaximo, *dev_escalaMinima, *dev_escalaMaxima, *dev_auxlerp;
	int size = sizePintado * sizeof(int);
	int sizefloat = sizePintado * sizeof(float);
	int sizevolt = sizeVoltaje * sizeof(float);
	int sizeElectros = numeroElectrodos * sizeof(int);

	voltajefinal = (float*)malloc(XSIZE*YSIZE*ZSIZE * sizeof(float));
	indexvolt = (int*)malloc(88055 * sizeof(int));
	hipMalloc(&dev_volt, XSIZE*YSIZE*ZSIZE * sizeof(float));
	//hipMemset(dev_volt, 0.9, XSIZE*YSIZE*ZSIZE * sizeof(float));
	hipMalloc((void **)&dev_xtam, sizeof(int));
	hipMalloc((void **)&dev_ytam, sizeof(int));
	hipMalloc((void **)&dev_ztam, sizeof(int));

	hipMalloc((void **)&dev_numeroElectrodos, sizeof(int));
	hipMalloc((void **)&dev_division, size);
	hipMalloc((void **)&dev_resto, size);
	hipMalloc((void **)&dev_indiceEgm, size);
	hipMalloc((void **)&dev_numeroTiempo, sizeof(int));
	hipMalloc((void **)&dev_contenidoEgm, sizefloat);
	hipMalloc((void **)&dev_indicex, sizeElectros);
	hipMalloc((void **)&dev_indicey, sizeElectros);
	hipMalloc((void **)&dev_valorMinimo, sizeof(float));
	hipMalloc((void **)&dev_valorMaximo, sizeof(float));
	hipMalloc((void **)&dev_escalaMinima, sizeof(float));
	hipMalloc((void **)&dev_escalaMaxima, sizeof(float));
	hipMalloc((void **)&dev_auxlerp, sizefloat);
	hipMalloc((void **)&dev_indexvolt, 88055 * sizeof(int));


	hipMemcpy(dev_xtam, &xtam, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_ytam, &ytam, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_ztam, &ztam, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dev_numeroElectrodos, &numeroElectrodos, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_numeroTiempo, &numeroTiempo, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_contenidoEgm, contenidoEgm, sizefloat, hipMemcpyHostToDevice);
	hipMemcpy(dev_indicex, indicex, sizeElectros, hipMemcpyHostToDevice);
	hipMemcpy(dev_indicey, indicey, sizeElectros, hipMemcpyHostToDevice);
	hipMemcpy(dev_valorMinimo, &valorMinimo, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_valorMaximo, &valorMaximo, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_escalaMinima, &escalaMinima, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_escalaMaxima, &escalaMaxima, sizeof(float), hipMemcpyHostToDevice);

	//dim3 threadsPerBlock(8, 8,4); //16x16x1 threads = 256 < 1024
	//dim3 numBlocks(ZSIZE / threadsPerBlock.x, YSIZE/ threadsPerBlock.y,XSIZE/threadsPerBlock.z);

	hipDeviceSynchronize();
	initvolt << < TAM2 / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(dev_xtam, dev_ytam, dev_ztam, dev_volt);
	hipDeviceSynchronize();
	//hipMemcpy(voltajefinal, dev_volt, XSIZE*YSIZE*ZSIZE * sizeof(float), hipMemcpyDeviceToHost);
	module << < TAM / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(dev_numeroElectrodos, dev_division, dev_resto, dev_indiceEgm, dev_numeroTiempo, dev_contenidoEgm, dev_indicex, dev_indicey, dev_valorMinimo, dev_valorMaximo, dev_auxlerp, dev_escalaMaxima, dev_escalaMinima, dev_volt, dev_xtam, dev_ytam, dev_ztam, dev_indexvolt);

	hipMemcpy(voltajefinal, dev_volt, XSIZE*YSIZE*ZSIZE * sizeof(float), hipMemcpyDeviceToHost);

	//hipMemcpy(indexvolt, dev_indexvolt, 88055 * sizeof(int), hipMemcpyDeviceToHost);
	//hipMemcpy(indiceegmlol, dev_indiceEgm, 88055 * sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(dev_xtam);
	hipFree(dev_ytam);
	hipFree(dev_ztam);
	hipFree(dev_volt);


	hipFree(dev_numeroElectrodos);
	hipFree(dev_division);
	hipFree(dev_numeroTiempo);
	hipFree(dev_resto);
	hipFree(dev_indiceEgm);
	hipFree(dev_contenidoEgm);
	hipFree(dev_indicex);
	hipFree(dev_indicey);
	hipFree(dev_valorMinimo);
	hipFree(dev_valorMaximo);
	hipFree(dev_escalaMinima);
	hipFree(dev_escalaMaxima);
	hipFree(dev_auxlerp);

	return voltajefinal;
}
