#include "hip/hip_runtime.h"
﻿#include "kernel.h"
#include <stdio.h>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include "kernel.h"


__global__ void module(int *indiceEgm, float *contentEgm, float * minValue, float * maxValue, float * auxLerp, float * maxScale, float * minScale, float* midScale, float * volt) {
	int pos = threadIdx.x + blockIdx.x * blockDim.x;
	if (pos < *indiceEgm) {
		auxLerp[pos] = (contentEgm[pos] + (-*minValue)) / (*maxValue + (-*minValue));
		volt[pos] = (1 - auxLerp[pos]) * *minScale + auxLerp[pos] * *maxScale;
	}
}




#define THREADS_PER_BLOCK 256

float * kernelArritmias(int sizeVoltaje, float * contenidoEgm, float valorMaximo, float valorMinimo, float escalaMaxima, float escalaMinima, float escalaMedia) {
	//Declaraci�n de variables
	float * voltajefinal;

	int *dev_indiceEgm;
	float  *dev_contenidoEgm, *dev_valorMinimo, *dev_valorMaximo, *dev_escalaMinima, *dev_escalaMaxima, *dev_auxlerp, *dev_escalaMedia, *dev_volt;

	int sizevolt = sizeVoltaje * sizeof(float);

	voltajefinal = (float*)malloc(sizevolt);


	hipMalloc((void **)&dev_indiceEgm, sizeof(int));
	hipMalloc((void **)&dev_contenidoEgm, sizevolt);
	hipMalloc((void **)&dev_valorMinimo, sizeof(float));
	hipMalloc((void **)&dev_valorMaximo, sizeof(float));
	hipMalloc((void **)&dev_escalaMedia, sizeof(float));
	hipMalloc((void **)&dev_escalaMinima, sizeof(float));
	hipMalloc((void **)&dev_escalaMaxima, sizeof(float));
	hipMalloc((void **)&dev_auxlerp, sizevolt);
	hipMalloc((void **)&dev_volt, sizevolt);

	hipMemcpy(dev_indiceEgm, &sizeVoltaje, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_contenidoEgm, contenidoEgm, sizevolt, hipMemcpyHostToDevice);
	hipMemcpy(dev_valorMinimo, &valorMinimo, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_valorMaximo, &valorMaximo, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_escalaMinima, &escalaMinima, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_escalaMaxima, &escalaMaxima, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_escalaMedia, &escalaMedia, sizeof(float), hipMemcpyHostToDevice);

	int nblocks = sizeVoltaje / THREADS_PER_BLOCK + sizeVoltaje % THREADS_PER_BLOCK;

	module << < nblocks, THREADS_PER_BLOCK >> >(dev_indiceEgm, dev_contenidoEgm, dev_valorMinimo, dev_valorMaximo, dev_auxlerp, dev_escalaMaxima, dev_escalaMinima, dev_escalaMedia, dev_volt);

	hipMemcpy(voltajefinal, dev_volt, sizevolt, hipMemcpyDeviceToHost);

	hipFree(dev_indiceEgm);
	hipFree(dev_contenidoEgm);
	hipFree(dev_valorMinimo);
	hipFree(dev_valorMaximo);
	hipFree(dev_escalaMinima);
	hipFree(dev_escalaMaxima);
	hipFree(dev_escalaMedia);
	hipFree(dev_auxlerp);
	hipFree(dev_volt);

	return voltajefinal;
}
