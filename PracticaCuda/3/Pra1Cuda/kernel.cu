#include "hip/hip_runtime.h"
#include "kernel.h"

#include <cstdio>

__global__ void convolution1Kernel(int *dst, int *src, int rows, int cols, int *filter) {
	// Convolucion en memoria global, similar a la convolucion en CPU
	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;
	if (posx > 0 && posy > 0 && posx < rows - 1 && posy < cols - 1) {
		for (int k = 0; k < 3; ++k) {
			for (int l = 0; l < 3; ++l) {
				dst[posy * cols + posx] += src[(posy + k - 1) * cols + (posx + l - 1)] * filter[k * 3 + l];
				//printf("Fuente = %i \n", src[(posy + k - 1) * cols + (posx + l - 1)]);
				//printf("Filtro = %i \n", filter[k * 3 + l]);

			}
		}
	}
	//printf("Destino = %i \n", dst[posy * cols + posx]);

}
__shared__ int smem[324];
__global__ void convolution2Kernel(int *dst, int *src, int rows, int cols, int *filter) {
	// Convolucion con memoria compartida
	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;
	//smem[threadIdx.y * (blockDim.x + 2) + threadIdx.x] = src[(idy - 1) * cols + (idx - 1)];
	if (threadIdx.x < 2 && threadIdx.y < 2) 
		smem[(threadIdx.y + blockDim.y) * (blockDim.x + 2) + threadIdx.x + blockDim.x] = src[(posy � 1 + blockDim.y) * cols + (posx � 1 + blockDim.x)];
}

void convolution(int *dst, int *src, int rows, int cols, int *filter) {
	size_t size = rows * cols * sizeof(int);
	size_t fsize = 3 * 3 * sizeof(int);

	int *d_src = NULL;
	int *d_filter = NULL;
	int *d_dst = NULL;

	//malloc en cuda para d_src, d_filter y d_dst
	hipMalloc(&d_src, size);
	hipMalloc(&d_filter, fsize);
	hipMalloc(&d_dst, size);

	//copiar los datos de src y filter a d_src y d_filter respectivamente
	hipMemcpy(d_src, src, size, hipMemcpyHostToDevice);
	hipMemcpy(d_filter, filter, fsize, hipMemcpyHostToDevice);


	//Poner d_dst a ceros
	hipMemset(d_dst, 0, size);

	//llamada al kernel
	dim3 threads(18, 18);
	dim3 blocks(1, 1);
	convolution2Kernel<<<blocks,threads>>>(d_dst, d_src, rows, cols, d_filter);

	//copiar los datos de vuelta. Los datos calculados y guardados en d_dst han de ser copiados a dst.
	hipMemcpy(dst, d_dst, size, hipMemcpyDeviceToHost);
	//liberar memoria que ya no se necesita
}
