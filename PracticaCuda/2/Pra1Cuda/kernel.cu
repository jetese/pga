#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include <cstdio>

//
__global__ void matrixAddPitch(float *a, float *b, float *c, int n,int m, size_t pitch) {
	// Escribir en c la suma de a y b
	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;
	if (posx < n && posy < m) {
		c[posx + posy *(pitch /sizeof(*a))] = a[posx + posy*(pitch / sizeof(*a))] + b[posx + posy*pitch / sizeof(*a)];
	}
};

__global__ void kernelAdd(float * a, float * b, float * c, int width, int height) {
	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;

	if (posx < width && posy < height) {
		c[posx + posy * width] = a[posx + posy * width] + b[posx + posy * width];
	}
}

__global__ void kernelAddPitch(float * a, float * b, float * c, int width, int height, size_t pitch) {
	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;

	if (posx < width && posy < height) {
		c[posx + posy * (pitch/sizeof(*a))] = a[posx + posy * (pitch / sizeof(*a))] + b[posx + posy * (pitch / sizeof(*a))];
	}
}
	

void vectorAdd(float *a, float *b, float *c, int n, int m) {
	////Declaraci�n de variables
	//float * d_a, *d_b, *d_c;
	//int width = n * sizeof(float);
	//int height = m * sizeof(float);
	//size_t size = n * m * sizeof(float);

	////Reservamos espacio en gpu
	//hipMalloc(&d_a, size);
	//hipMalloc(&d_b, size);
	//hipMalloc(&d_c, size);

	////Copiamos las variables de CPU a GPU
	//hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	//hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	////N�mero de threads y de bloques
	//dim3 threads = (4, 4);
	//dim3 blocks = ((n - 1) / threads.x + 1, (m - 1) / threads.y + 1);

	////LLamada a la funcion
	//kernelAdd << <blocks, threads >> > (d_a, d_b, d_c, n, m);

	////Copiamos datos de GPU -> CPU
	//hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	////Liberamos memoria
	//hipFree(d_a);
	//hipFree(d_b);
	//hipFree(d_c);

	//Declaraci�n de variables
	float * d_a, *d_b, *d_c;
	int width = n ;
	int height = m;
	size_t size = n * m * sizeof(float);
	size_t pitch;

	//Reserva de memoria GPU
	//hipMallocPitch(&d_a, &pitch, width, height);
	//hipMallocPitch(&d_b, &pitch, width, height);
	//hipMallocPitch(&d_c, &pitch, width, height);
	hipMallocPitch(&d_a, &pitch, sizeof(float)* width, height);
	hipMallocPitch(&d_b, &pitch, sizeof(float)* width, height);
	hipMallocPitch(&d_c, &pitch, sizeof(float)* width, height);

	// Copia de datos CPU -> GPU
	//hipMemcpy2D(d_a, pitch, a, width, width, height, hipMemcpyHostToDevice);
	//hipMemcpy2D(d_b, pitch, b, width, width, height, hipMemcpyHostToDevice);
	hipMemcpy2D(d_a, pitch, a, sizeof(float)* width, sizeof(float)* width, height, hipMemcpyHostToDevice);
	hipMemcpy2D(d_b, pitch, b, sizeof(float)* width, sizeof(float)* width, height, hipMemcpyHostToDevice);

	//Threads y bloques
	dim3 threads (4, 4);
	dim3 blocks ((m - 1) / threads.x + 1, (n - 1) / threads.y + 1);
	//dim3 threads(4, 4);
	//dim3 blocks((m - 1) / threads.x + 1, (n - 1) / threads.y + 1);

	//LLamada a la funci�n
	//kernelAddPitch << <blocks, threads >> > (d_a, d_b, d_c, n, m, pitch);
	matrixAddPitch << < blocks, threads >> >(d_a, d_b, d_c, n, m, pitch);

	// Copia de GPU -> CPU
	//hipMemcpy2D(c, width * sizeof(float), d_c, pitch, width * sizeof(float), height, hipMemcpyDeviceToHost);
	hipMemcpy2D(c, sizeof(float)*width, d_c, pitch, sizeof(float)* width, height, hipMemcpyDeviceToHost);

	hipFree(d_a);
};
//size_t size = n *m * sizeof(float);
//size_t size2 = n*m;

//float *d_a = NULL;
//float *d_b = NULL;
//float *d_c = NULL;
//int dev_n, dev_m;
//size_t d_pitch = NULL;
//size_t pitch = NULL;
//size_t width = n;
//size_t height = m;


////Cambiar variables, los pitch no pueden ser los mismos
//hipMallocPitch(&d_a, &pitch, sizeof(float)* width, height);
//hipMallocPitch(&d_b, &pitch, sizeof(float)* width, height);
//hipMallocPitch(&d_c, &pitch, sizeof(float)* width, height);//Me devuelve el ancho de fila que ha reservado realmente en el pitch ( lo devuelve en bytes)

//hipMemcpy2D(d_a, pitch, a, sizeof(float)* width, sizeof(float)* width, height, hipMemcpyHostToDevice);
//hipMemcpy2D(d_b, pitch, b, sizeof(float)* width, sizeof(float)* width, height, hipMemcpyHostToDevice);
//
////llamada al kernel
//dim3 threads(4, 4);
////dim3 blocks((size2 + threads.x - 1) / threads.x, (size2 + threads.y - 1)/threads.y, 1);
////(col-1)/thread.x +1 , (row-1)/thread.y +1
//dim3 blocks((m - 1) / threads.x + 1, (n - 1) / threads.y + 1);
////matrixAdd << < blocks, threads >> >(d_a, d_b, d_c, n, m);
//matrixAddPitch << < blocks, threads >> >(d_a, d_b, d_c, n, m, pitch);
//
////copiar los datos de vuelta. Los datos calculados y guardados en d_c han de ser copiados a c.
////hipMemcpy2D(c,d_c , size, hipMemcpyDeviceToHost);
//hipMemcpy2D(c, sizeof(float)*width, d_c, pitch, sizeof(float)* width, height, hipMemcpyDeviceToHost);
////liberar memoria que ya no se necesita
//hipFree(d_a);
//hipFree(d_b);
//hipFree(d_c);