#include "hip/hip_runtime.h"
#include "kernel.h"

#include <cstdio>

__global__ void convolution1Kernel(int *dst, int *src, int rows, int cols, int *filter) {
	// Convolucion en memoria global, similar a la convolucion en CPU
}

__global__ void convolution2Kernel(int *dst, int *src, int rows, int cols, int *filter) {
	// Convolucion con memoria compartida
}

void convolution(int *dst, int *src, int rows, int cols, int *filter) {
	size_t size = rows * cols * sizeof(int);
	size_t fsize = 3 * 3 * sizeof(int);

	int *d_src = NULL;
	int *d_filter = NULL;
	int *d_dst = NULL;

	//malloc en cuda para d_src, d_filter y d_dst

	//copiar los datos de src y filter a d_src y d_filter respectivamente

	//Poner d_dst a ceros
	hipMemset(d_dst, 0, size);

	//llamada al kernel

	//copiar los datos de vuelta. Los datos calculados y guardados en d_dst han de ser copiados a dst.

	//liberar memoria que ya no se necesita
}
