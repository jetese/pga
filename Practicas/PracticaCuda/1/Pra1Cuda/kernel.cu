#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void vectorAddKernel(float * da, float * db, float * dc, int n) {
	int pos = threadIdx.x + blockDim.x * blockIdx.x;
	if (threadIdx.x < n) {
		dc[pos] = da[pos] + db[pos];
	}
}

void vectorAdd(float *a, float *b, float *c, int n) {
	
	float * dev_a ,*dev_b ,* dev_c ;
	size_t size = n * sizeof(float);

	hipMalloc(&dev_a, size);
	hipMalloc(&dev_b, size);
	hipMalloc(&dev_c, size);

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	vectorAddKernel << <2, n/2 >> > (dev_a, dev_b, dev_c, n);

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

};
