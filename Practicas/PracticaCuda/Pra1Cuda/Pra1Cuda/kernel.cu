#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void vectorAddKernel(float *a, float *b, float *c, int n) {
	// Escribir en c la suma de a y b
	int pos = threadIdx.x + blockIdx.x * blockDim.x;
	if(pos < n)
	c[pos] = a[pos] + b[pos];
};


void vectorAdd(float *a, float *b, float *c, int n) {
	size_t size = n * sizeof(float);

	float *d_a = NULL;
	float *d_b = NULL;
	float *d_c = NULL;

	//malloc en cuda para d_a, d_b y d_c
	hipMalloc(&d_a,size);
	hipMalloc(&d_b,size);
	hipMalloc(&d_c,size);
	//copiar los datos de a y b a d_a y d_b respectivamente
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	//llamada al kernel
	vectorAddKernel<<<1, n >>>(d_a, d_b, d_c, n);

	//copiar los datos de vuelta. Los datos calculados y guardados en d_c han de ser copiados a c.
	hipMemcpy(c,d_c , size, hipMemcpyDeviceToHost);
	//liberar memoria que ya no se necesita
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
};
