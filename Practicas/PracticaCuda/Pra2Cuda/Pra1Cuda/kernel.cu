#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#include <cstdio>

__global__ void matrixAdd(float *a, float *b, float *c, int n, int m) {
	// Escribir en c la suma de a y b
	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;
	if (posx < n && posy < m) {
		c[posx + posy *n] = a[posx + posy*n] + b[posx + posy*n];
		printf("posx %d \n", posx);
		printf("posy %d \n", posy);
	}
};

__global__ void matrixAddPitch(float *a, float *b, float *c, int n,int m, size_t pitch) {
	// Escribir en c la suma de a y b
	int posx = threadIdx.x + blockIdx.x * blockDim.x;
	int posy = threadIdx.y + blockIdx.y * blockDim.y;
	if (posx < n && posy < m) {
		c[posx + posy *(pitch /sizeof(*a))] = a[posx + posy*(pitch / sizeof(*a))] + b[posx + posy*pitch / sizeof(*a)];
	}
};
	

void vectorAdd(float *a, float *b, float *c, int n, int m) {
	size_t size = n *m* sizeof(float);
	size_t size2 = n*m;
	//byte a;
	float *d_a = NULL;
	float *d_b = NULL;
	float *d_c = NULL;
	int dev_n,dev_m;
	size_t d_pitch=NULL;
	size_t pitch=NULL;
	size_t width=n;
	size_t height=m;
	//malloc en cuda para d_a, d_b y d_c
	/*hipMalloc(&d_a,size);
	hipMalloc(&d_b,size);
	hipMalloc(&d_c,size);*/
	//Cambiar variables, los pitch no pueden ser los mismos
	hipMallocPitch(&d_a, &pitch, sizeof(float)* width, height);
	hipMallocPitch(&d_b, &pitch, sizeof(float)* width, height);
	hipMallocPitch(&d_c, &pitch, sizeof(float)* width, height);//Me devuelve el ancho de fila que ha reservado realmente en el pitch ( lo devuelve en bytes)

	//copiar los datos de a y b a d_a y d_b respectivamente
	//hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	//hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy2D(d_a, pitch, a, sizeof(float)* width, sizeof(float)* width, height, hipMemcpyHostToDevice);
	hipMemcpy2D(d_b, pitch, b, sizeof(float)* width, sizeof(float)* width, height, hipMemcpyHostToDevice);

	//llamada al kernel
	dim3 threads(4, 4);
	//dim3 blocks((size2 + threads.x - 1) / threads.x, (size2 + threads.y - 1)/threads.y, 1);
	//(col-1)/thread.x +1 , (row-1)/thread.y +1
	dim3 blocks((m - 1) / threads.x + 1, (n - 1) / threads.y + 1);
	//matrixAdd << < blocks, threads >> >(d_a, d_b, d_c, n, m);
	matrixAddPitch << < blocks, threads >> >(d_a, d_b, d_c,n,m, pitch);

	//copiar los datos de vuelta. Los datos calculados y guardados en d_c han de ser copiados a c.
	//hipMemcpy2D(c,d_c , size, hipMemcpyDeviceToHost);
	hipMemcpy2D(c, sizeof(float)*width, d_c, pitch, sizeof(float)* width, height, hipMemcpyDeviceToHost);
	//liberar memoria que ya no se necesita
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
};
